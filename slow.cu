#include <hip/hip_runtime.h>
#include <math.h>

extern "C" {

__global__ void compute_attention(
    const float *q, const float *k, const float *mask,
    float *scores, int seq_len, int depth) {

    int i = blockIdx.x * blockDim.x + threadIdx.x; // Query index
    int j = blockIdx.y * blockDim.y + threadIdx.y; // Key index

    if (i < seq_len && j < seq_len) {
        float score = 0.0f;
        for (int d = 0; d < depth; ++d) {
            score += q[i * depth + d] * k[j * depth + d];
        }
        score /= sqrtf((float)depth);
        score += mask[i * seq_len + j];
        scores[i * seq_len + j] = score;
    }
}

__global__ void softmax_kernel(float *scores, int seq_len) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < seq_len) {
        float max_score = -1e20f;
        for (int j = 0; j < seq_len; ++j) {
            float val = scores[i * seq_len + j];
            if (val > max_score) max_score = val;
        }
        float sum_exp = 0.0f;
        for (int j = 0; j < seq_len; ++j) {
            float val = expf(scores[i * seq_len + j] - max_score);
            scores[i * seq_len + j] = val;
            sum_exp += val;
        }
        for (int j = 0; j < seq_len; ++j) {
            scores[i * seq_len + j] /= sum_exp;
        }
    }
}

__global__ void compute_output(
    const float *scores, const float *v, float *output,
    int seq_len, int depth) {

    int i = blockIdx.x * blockDim.x + threadIdx.x; // Query index
    int d = blockIdx.y * blockDim.y + threadIdx.y; // Depth index

    if (i < seq_len && d < depth) {
        float result = 0.0f;
        for (int j = 0; j < seq_len; ++j) {
            result += scores[i * seq_len + j] * v[j * depth + d];
        }
        output[i * depth + d] = result;
    }
}

void attention(
    const float *q, const float *k, const float *v, const float *mask,
    float *output, int seq_len, int depth) {

    float *d_q, *d_k, *d_v, *d_mask, *d_scores, *d_output;

    size_t size_qkv = sizeof(float) * seq_len * depth;
    size_t size_scores = sizeof(float) * seq_len * seq_len;

    hipMalloc((void**)&d_q, size_qkv);
    hipMalloc((void**)&d_k, size_qkv);
    hipMalloc((void**)&d_v, size_qkv);
    hipMalloc((void**)&d_mask, size_scores);
    hipMalloc((void**)&d_scores, size_scores);
    hipMalloc((void**)&d_output, size_qkv);

    hipMemcpy(d_q, q, size_qkv, hipMemcpyHostToDevice);
    hipMemcpy(d_k, k, size_qkv, hipMemcpyHostToDevice);
    hipMemcpy(d_v, v, size_qkv, hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, size_scores, hipMemcpyHostToDevice);

    dim3 blockDim(32, 32);
    dim3 gridDim((seq_len + blockDim.x - 1) / blockDim.x, (seq_len + blockDim.y - 1) / blockDim.y);

    compute_attention<<<gridDim, blockDim>>>(d_q, d_k, d_mask, d_scores, seq_len, depth);

    int threads = 32;
    int blocks = (seq_len + threads - 1) / threads;

    softmax_kernel<<<blocks, threads>>>(d_scores, seq_len);

    dim3 blockDim2(32, 32);
    dim3 gridDim2((seq_len + blockDim2.x - 1) / blockDim2.x, (depth + blockDim2.y - 1) / blockDim2.y);

    compute_output<<<gridDim2, blockDim2>>>(d_scores, d_v, d_output, seq_len, depth);

    // Copy result back to host
    hipMemcpy(output, d_output, size_qkv, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_q);
    hipFree(d_k);
    hipFree(d_v);
    hipFree(d_mask);
    hipFree(d_scores);
    hipFree(d_output);
}

}
